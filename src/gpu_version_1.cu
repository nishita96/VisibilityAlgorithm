
// #include "versions.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>

#include <iostream>
// #include "ofMain.h"

#define PI 3.14

class ofVec2f {
public:
    float x;
    float y;
    
    ofVec2f(float _x, float _y) {
        x = _x;
        y = _y;
    }

    __device__ ofVec2f(float _x, float _y, int c) {
        x = _x;
        y = _y;
    }

    void set(float _x, float _y) {
        x = _x;
        y = _y;
    }

    float angle(ofVec2f v1) {
        return 0;
    }
    float length() {
        return 0;
    }

     ofVec2f getNormalized() {
        ofVec2f new_vec(x, y);
        return new_vec;
    }

     ofVec2f operator*(float scale) {
        ofVec2f vec(1,1);
        vec.x = this->x * scale;
        vec.y = this->y * scale;
        return vec;
    }
};

// __device__ ofVec2f* create_ofvec_obj(int x, int y) {
//     ofVec2f* vec = new ofVec2f(x, y, 0);
//     return vec;
// }

using namespace std;
#define BLOCK_SIZE 512 //@@ You can change this

class vray {
public:
    vray(float _theta, ofVec2f _unitVec, float _r, float _l) :
        theta(_theta), unitVec(_unitVec.x, _unitVec.y), r(_r), l(_l) {}
    // vray(float _theta, float _r, float _l) :
    //     theta(_theta), r(_r), l(_l) {}
    
    float theta;
    ofVec2f unitVec;
    float r;
    float l;

};

class segment {
public:
     segment(ofVec2f e0, ofVec2f e1) :
        p0(e0.x, e0.y), p1(e1.x, e1.y) {}

    ofVec2f p0;
    ofVec2f p1;
    bool isValid = false;
    
     int signum(float f);
    __device__ void translateToQ(ofVec2f pointQ){
        printf("inside translateToQ\n");
        p0.set(p0.x - pointQ.x, -(p0.y - pointQ.y)); // because display needs oroginal coordinates but geomterically the y direction is opposite
        p1.set(p1.x - pointQ.x, -(p1.y - pointQ.y));
    }
     bool possibleIntersectionTestXAxis();
     ofVec2f splitSegmentInto2();
     ofVec2f intersectionWithGivenSegment(segment other);
     bool collinearWithQ(); // q is always 0 after translation
     vector<vray> generateVray(segment seg);
};



// void segment::translateToQ(ofVec2f pointQ){
//     p0.set(p0.x - pointQ.x, -(p0.y - pointQ.y)); // because display needs oroginal coordinates but geomterically the y direction is opposite
//     p1.set(p1.x - pointQ.x, -(p1.y - pointQ.y));
// }


int segment::signum(float f) {
   if (f > 0) return 1;
   if (f < 0) return -1;
   return 0;
}


bool segment::possibleIntersectionTestXAxis(){
    // TODO: Implement A Method To Find The Intersection Between 2 Axis Aligned Bounding Boxes
    if(p0.x < 0 && p1.x < 0){
        return false;
    }
    if(p0.y < 0 && p1.y < 0){
        return false;
    }
    if(p0.y > 0 && p1.y > 0){
        return false;
    }
    return true;
  }


  ofVec2f segment::splitSegmentInto2(){
    // TODO: Implement A Fast Method To Find The Edge Intersection Point.
    // Should return the intersection point or null, if no intersection exists.
    //  Care should be taken to make the implementation CORRECT, but SPEED MATTERS.
     
    segment other(ofVec2f(0.0f, 0.0f), ofVec2f(1500.0f, 0.0f)); // x axis as segment
    //do the p1 - p0 and use that so that number of multiplication is less
    float p0x = other.p0.x;
    float p0y = other.p0.y;
    float p1x = other.p1.x;
    float p1y = other.p1.y;
    float p2x = p0.x;
    float p2y = p0.y;
    float p3x = p1.x;
    float p3y = p1.y;
    float s = (p2y*p1x - p0y*p1x - p2y*p0x + p0y*p0x - p2x*p1y + p0x*p1y + p2x*p0y - p0x*p0y)/(p3x*p1y - p2x*p1y - p3x*p0y + p2x*p0y - p3y*p1x + p2y*p1x + p3y*p0x - p2y*p0x);
    if(0 < s && s < 1){ // does not take the end points
        float t = (p2x + ((p3x - p2x) * s) - p0x)/(p1x - p0x);
        if(0 < t && t < 1){ // does not take the end points
             float pointX = p0x + ((p1x - p0x) * t);
             float pointY = p0y + ((p1y - p0y) * t);
             return ofVec2f(pointX, pointY);
        }
    }
    return ofVec2f(-1.0f, 0.0f); // if no intersection THEN returned point has x<0
}


ofVec2f segment::intersectionWithGivenSegment(segment other){
    // TODO: Implement A Fast Method To Find The Edge Intersection Point.
    // Should return the intersection point or null, if no intersection exists.
    //  Care should be taken to make the implementation CORRECT, but SPEED MATTERS.
     
//    segment other(ofVec2f(0.0f, 0.0f), ofVec2f(1500.0f, 0.0f)); // x axis as segment
    //do the p1 - p0 and use that so that number of multiplication is less
    float p0x = other.p0.x;
    float p0y = other.p0.y;
    float p1x = other.p1.x;
    float p1y = other.p1.y;
    float p2x = p0.x;
    float p2y = p0.y;
    float p3x = p1.x;
    float p3y = p1.y;
    float s = (p2y*p1x - p0y*p1x - p2y*p0x + p0y*p0x - p2x*p1y + p0x*p1y + p2x*p0y - p0x*p0y)/(p3x*p1y - p2x*p1y - p3x*p0y + p2x*p0y - p3y*p1x + p2y*p1x + p3y*p0x - p2y*p0x);
    if(0 < s && s < 1){ // does not take the end points
        float t = (p2x + ((p3x - p2x) * s) - p0x)/(p1x - p0x);
        if(0 < t && t < 1){ // does not take the end points
             float pointX = p0x + ((p1x - p0x) * t);
             float pointY = p0y + ((p1y - p0y) * t);
             return ofVec2f(pointX, pointY);
        }
    }
    return ofVec2f(0.0f, 0.0f); // if no intersection THEN returned point has x<0, TODO how to pass null
}


bool segment::collinearWithQ(){
    return abs(p0.x * p1.y - p0.y * p1.x)/2;
}


vector<vray> segment::generateVray(segment seg){
    vector<vray> bothVray;
    ofVec2f xAxisVec(1.0,0.0);
    float infinity = 9999.0f;
    
    if((seg.p0.x * seg.p1.y - seg.p0.y * seg.p1.x) < 0){ // p0 X p1
        // to ensure p0 theta < p1 theta
        swap(seg.p0, seg.p1);
    }
    float theta0 = xAxisVec.angle(seg.p0);
    float theta1;
    if(xAxisVec.angle(seg.p1) == 0.0f){
        theta1 = 360.0;
    }
    else{
        theta1 = xAxisVec.angle(seg.p1);
    }
    theta0 = theta0 < 0? theta0 + 360.0f : theta0;
    theta1 = theta1 < 0? theta1 + 360.0f : theta1;
    bothVray.push_back(vray(theta0, seg.p0.getNormalized(), infinity, seg.p0.length()));
    bothVray.push_back(vray(theta1, seg.p1.getNormalized(), seg.p1.length(), infinity));
    
    return bothVray;
}

__global__ void preprocess_in_parallel(segment* input, int size, int x, int y, segment* output) {
 
    int tid = threadIdx.x;
    if(tid < size) {
        // ofVec2f* q = create_ofvec_obj(x, y);
        ofVec2f q(x, y, 0);
        printf("sdfs\n");
        printf("Output: input[tid]: %f\n", input[tid].p1.x);
        input[tid].translateToQ(q);
        
        // if(input[tid].collinearWithQ() != 0.0f){
        //     // input[tid].erase(it);
        //     if(input[tid].possibleIntersectionTestXAxis()) {
        //         ofVec2f splitPoint = input[tid].splitSegmentInto2();
        //         if(splitPoint.x != -1.0f){ 
        //             output[2 * tid] = segment(input[tid].p0, splitPoint);
        //             output[2 * tid + 1] = segment(input[tid].p1, splitPoint);
        //             output[2 * tid].isValid = true;
        //             output[2 * tid + 1].isValid = true;
        //         } else {
        //             output[2 * tid] = segment(input[tid].p0, input[tid].p1);
        //             output[2 * tid].isValid = true;
        //         }
        //     } else {
        //         output[2 * tid] = segment(input[tid].p0, input[tid].p1);
        //         output[2 * tid].isValid = true;
        //     }
        // }
    }
}

class GPU_V1 {
    float infinity = 9999.0f;
    vector<segment> check_intersections(vector<segment>& segments) {
        vector<segment> listSegmentsCopy;
        for (auto seg: segments){
            if(seg.possibleIntersectionTestXAxis()){
                ofVec2f splitPoint = seg.splitSegmentInto2();
                if(splitPoint.x != -1.0f){ // there is intersection, HENCE split it in 2 segments
                    // TODO check if you need to check which has smaller angle
                    listSegmentsCopy.push_back(segment(seg.p0, splitPoint));
                    listSegmentsCopy.push_back(segment(seg.p1, splitPoint));
                }
                else{
                    listSegmentsCopy.push_back(segment(seg.p0, seg.p1));
                }
            }
            else{
                listSegmentsCopy.push_back(segment(seg.p0, seg.p1));
            }
        }
        return listSegmentsCopy;
    }

    vector<vray> mergeVrays(vector<vray> l1, vector<vray> l2){
        vector<vray> l;
        
        int n = l1.size() + l2.size();
        int n1 = l1.size();
        int n2 = l2.size();
    //    cout <<  "\n size l1:" << l1.size() << " l2:" << l2.size() << " l:" << l.size();
    //    cout << "\n l1";
    //    printAllVrays(l1);
    //    cout << "\n l2";
    //    printAllVrays(l2);
        
        int i = 0; // diff from paper
        int i1 = 0;
        int i2 = 0;
        int k = 0;
        int t = 0;
        
        vector<vray> lk;
        vector<vray> lt;
        int ik=0;
        int it=0;
        

        while(i < n){
            if(i2 >= n2){
                k = 1;
                ik = i1;
                lk = l1;
                t = 2;
                it = i2;
                lt = l2;
            }
            else if(i1 >= n1){
                k = 2;
                ik = i2;
                lk = l2;
                t = 1;
                it = i1;
                lt = l1;
            }
            else if(l1.at(i1).theta <= l2.at(i2).theta){
                k = 1;
                ik = i1;
                lk = l1;
                t = 2;
                it = i2;
                lt = l2;
            }
            else{
                k = 2;
                ik = i2;
                lk = l2;
                t = 1;
                it = i1;
                lt = l1;
            }

            l.push_back(lk.at(ik));

            if(0 < it && it < lt.size() && lt.at(it).r < infinity ){
                segment s = segment(lt.at(it-1).unitVec * lt.at(it-1).l, lt.at(it).unitVec * lt.at(it).r);
                float thetaRad = l.at(i).theta * PI / 180.0;
                segment other(ofVec2f(0.0f, 0.0f), ofVec2f(1500.0f * cos(thetaRad), 1500.0f * sin(thetaRad))); // x axis as segment
                ofVec2f p = s.intersectionWithGivenSegment(other);
    //            if(p != ofVec2f(0.0f, 0.0f)){
    //                cout << "\n point" << p.x << " " << p.y ;
    //            }
    //            else{
    //                cout << "\n it is null - did not find an intersection ";
    //            }
                

                l.at(i).l = min(l.at(i).l, p.length());
                l.at(i).r = min(l.at(i).r, p.length());
            }

            if(k == 1){
                i1 = i1 + 1; // can assign k = i1 or i2 initially based on condition so that we  have to chec value of k again n again
            }
            else{
                i2 = i2 + 1;
            }
            i = i + 1;
        }
        
    //    cout << "\n l";
    //    printAllVrays(l);
        return l;
    }

    vector<vray> merge(vector<segment>& listSegmentsCopy) {
        
        
        int minValueR = 9999.0f;
        vector<vray> vrayForMerge;
        vrayForMerge.push_back(listSegmentsCopy.at(0).generateVray(listSegmentsCopy.at(0)).at(0));
        vrayForMerge.push_back(listSegmentsCopy.at(0).generateVray(listSegmentsCopy.at(0)).at(1));
        if(vrayForMerge.at(0).theta == 360.0f && vrayForMerge.at(0).r < minValueR){
            minValueR = vrayForMerge.at(0).r;
        }
        if(vrayForMerge.at(1).theta == 360.0f && vrayForMerge.at(1).r < minValueR){
            minValueR = vrayForMerge.at(1).r;
        }
        vector<vray> vrayNewPair;
        for (int i=1; i<listSegmentsCopy.size(); i++){
            segment seg = listSegmentsCopy.at(i);
            vrayNewPair.clear();
            vrayNewPair.push_back(seg.generateVray(seg).at(0));
            vrayNewPair.push_back(seg.generateVray(seg).at(1));
            if(vrayNewPair.at(0).theta == 360.0f && vrayNewPair.at(0).r < minValueR){
                minValueR = vrayNewPair.at(0).r;
            }
            if(vrayNewPair.at(1).theta == 360.0f && vrayNewPair.at(1).r < minValueR){
                minValueR = vrayNewPair.at(1).r;
            }
            vrayForMerge = mergeVrays(vrayForMerge, vrayNewPair);
        }
        vrayForMerge.push_back(vray(360.0, ofVec2f(1.0f, 0.0f), minValueR, infinity));
        return vrayForMerge;
    }

    vector<segment> preprocess(vector<segment>& segments, ofVec2f& q) {
            
        hipError_t err = hipSuccess;
        segment *d_segments;
        hipMalloc(&d_segments, segments.size() * sizeof(segment));
        err = hipMemcpy(d_segments, segments.data(), segments.size() * sizeof(segment), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate d_segments (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        segment *d_output_segments;
        hipMalloc(&d_output_segments, 2 * segments.size() * sizeof(segment));

        // ofVec2f *d_q;
        // cudaMalloc((void**)d_q, sizeof(ofVec2f));
        // err = cudaMemcpy(d_q, &q, sizeof(ofVec2f), cudaMemcpyHostToDevice);
        // if (err != cudaSuccess)
        // {
        //     fprintf(stderr, "Failed to allocate d_q (error code %s)!\n", cudaGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }


        int threadsPerBlock = 100;
        int blocksPerGrid = 1;
        // preprocess_in_parallel <<<blocksPerGrid, threadsPerBlock>>> (d_segments, segments.size(), q.x, q.y, d_output_segments);
        // cudaDeviceSynchronize();
        // err = cudaGetLastError();
        // if (err != cudaSuccess)
        // {
        //     fprintf(stderr, "Failed to launch preprocess_in_parallel kernel (error code %s)!\n", cudaGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }
        // vector<segment> preprocessed_segments;
        // err = cudaMemcpy(preprocessed_segments.data(), d_output_segments, 2 * segments.size() * sizeof(segment), cudaMemcpyDeviceToHost);
        // // cudaFree(d_output_segments);
        // // cudaFree(d_segments);
        // vector<segment> filtered_segments;
        // for(auto ps : preprocessed_segments) {
        //     if(ps.isValid) {
        //         filtered_segments.push_back(ps);
        //     }
        // }
        vector<segment> filtered_segments;
        return filtered_segments;

    }

    public:
    virtual vector<vray> process_segments(vector<segment> segments, ofVec2f& q) {
        // start = 0;
        std::cout<<"Starting Process"<<std::endl;
        vector<segment> updated_segments = this->preprocess(segments, q);
        std::cout<<"Preprocess complete: "<< updated_segments.size() <<std::endl;
        vector<vray> merged_vrays;
        // vector<vray> merged_vrays = this->merge(updated_segments);
        // std::cout<<"Ending Process"<<std::endl;
        // end = 0;
        return merged_vrays;
    }
};


int main() {
    vector<segment> listSegments = {
        segment(ofVec2f(600.0f, 550.0f), ofVec2f(650.0f, 400.0f)),  // right small, line to split at 0 degree
        segment(ofVec2f(100.0f, 500.0f), ofVec2f(350.0f, 300.0f)),  // connected pair above
        segment(ofVec2f(550.0f, 700.0f), ofVec2f(100.0f, 500.0f)),  // connected pair bottom
        segment(ofVec2f(600.0f, 600.0f), ofVec2f(800.0f, 150.0f)),  // right side long
        segment(ofVec2f(200.0f, 300.0f), ofVec2f(700.0f, 100.0f)),  // top horizontal

//        segment(ofVec2f(800.0f, 500.0f), ofVec2f(800.0f, 500.01f)),   // the line for ending
        segment(ofVec2f(450.0f, 450.0f), ofVec2f(400.0f, 400.0f))   // the collinear line
    };
    ofVec2f pointQ(500,400);
    GPU_V1 *v1 = new GPU_V1();
    v1->process_segments(listSegments, pointQ);
    return 0;
}

