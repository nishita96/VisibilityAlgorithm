#include "hip/hip_runtime.h"

// #include "versions.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <iostream>
// #include "ofMain.h"
#include <ctime>
#include <cmath>

#define PI 3.14

class ofVec2f {
public:
    float x;
    float y;
    
    ofVec2f(float _x, float _y) {
        x = _x;
        y = _y;
    }

    __device__ ofVec2f(float _x, float _y, int c) {
        x = _x;
        y = _y;
    }

    __device__ ofVec2f(ofVec2f of, int c) {
        x = of.x;
        y = of.y;
    }

    __device__ void set(float _x, float _y) {
        x = _x;
        y = _y;
    }

    float angle(ofVec2f v1) {
        float dotProduct = 0.0;
        float normA = 0.0;
        float normB = 0.0;

        dotProduct = x*v1.x + y*v1.y; // += a[i] * b[i];
        normA = x*x + y*y; // += a[i] * a[i];
        normB = v1.x*v1.x + v1.y*v1.y; // += b[i] * b[i];


        normA = sqrt(normA);
        normB = sqrt(normB);

        float cosTheta = dotProduct / (normA * normB);
        float theta = acos(cosTheta);

        float crossProduct = x*v1.y + y*v1.x; //a[0]*b[1] - a[1]*b[0]; 
        if (crossProduct < 0) {
            theta = -theta; // adjust theta if cross product is negative
        }
        theta = theta * 180 / M_PI;
        return theta;
    }
    
    float length() {
        return sqrt(x*x + y*y);
    }

    __device__ float lengthGpu() {
        return sqrt(x*x + y*y);
    }

    ofVec2f getNormalized() {
        ofVec2f new_vec(x, y);
        return new_vec;
    }

    __device__ ofVec2f operator*(float scale) {
        ofVec2f vec(1,1);
        vec.x = this->x * scale;
        vec.y = this->y * scale;
        return vec;
    }

};

// __device__ ofVec2f* create_ofvec_obj(int x, int y) {
//     ofVec2f* vec = new ofVec2f(x, y, 0);
//     return vec;
// }

using namespace std;
#define BLOCK_SIZE 512 //@@ You can change this

class vray {
public:
    vray(float _theta, ofVec2f _unitVec, float _r, float _l) :
        theta(_theta), unitVec(_unitVec.x, _unitVec.y), r(_r), l(_l) {}
    __device__ vray(float _theta, ofVec2f _unitVec, float _r, float _l, int n) :
        theta(_theta), unitVec(_unitVec.x, _unitVec.y), r(_r), l(_l) {}
        
    // vray(float _theta, float _r, float _l) :
    //     theta(_theta), r(_r), l(_l) {}
    
    float theta;
    ofVec2f unitVec;
    float r;
    float l;

};

class segment {
public:
     segment(ofVec2f e0, ofVec2f e1) :
        p0(e0.x, e0.y), p1(e1.x, e1.y) {}

    __device__ segment(ofVec2f e0, ofVec2f e1, int c) :
    p0(e0.x, e0.y, 0), p1(e1.x, e1.y, 0) {}

    ofVec2f p0;
    ofVec2f p1;
    bool isValid = false;
    
     int signum(float f);
    __device__ void translateToQ(ofVec2f pointQ){
        // printf("inside translateToQ\n");
        p0.set(p0.x - pointQ.x, -(p0.y - pointQ.y)); // because display needs oroginal coordinates but geomterically the y direction is opposite
        p1.set(p1.x - pointQ.x, -(p1.y - pointQ.y));
    }
    __device__ bool possibleIntersectionTestXAxis();
    __device__ ofVec2f splitSegmentInto2();
    __device__ ofVec2f intersectionWithGivenSegment(segment other);
     __device__ bool collinearWithQ(); // q is always 0 after translation
    vector<vray> generateVray(segment seg);
};



// void segment::translateToQ(ofVec2f pointQ){
//     p0.set(p0.x - pointQ.x, -(p0.y - pointQ.y)); // because display needs oroginal coordinates but geomterically the y direction is opposite
//     p1.set(p1.x - pointQ.x, -(p1.y - pointQ.y));
// }


int segment::signum(float f) {
   if (f > 0) return 1;
   if (f < 0) return -1;
   return 0;
}


__device__ bool segment::possibleIntersectionTestXAxis(){
    // TODO: Implement A Method To Find The Intersection Between 2 Axis Aligned Bounding Boxes
    if(p0.x < 0 && p1.x < 0){
        return false;
    }
    if(p0.y < 0 && p1.y < 0){
        return false;
    }
    if(p0.y > 0 && p1.y > 0){
        return false;
    }
    return true;
  }


  __device__ ofVec2f segment::splitSegmentInto2(){
    // TODO: Implement A Fast Method To Find The Edge Intersection Point.
    // Should return the intersection point or null, if no intersection exists.
    //  Care should be taken to make the implementation CORRECT, but SPEED MATTERS.
     
    segment other(ofVec2f(0.0f, 0.0f, 0), ofVec2f(1500.0f, 0.0f, 0), 0); // x axis as segment
    //do the p1 - p0 and use that so that number of multiplication is less
    float p0x = other.p0.x;
    float p0y = other.p0.y;
    float p1x = other.p1.x;
    float p1y = other.p1.y;
    float p2x = p0.x;
    float p2y = p0.y;
    float p3x = p1.x;
    float p3y = p1.y;
    float s = (p2y*p1x - p0y*p1x - p2y*p0x + p0y*p0x - p2x*p1y + p0x*p1y + p2x*p0y - p0x*p0y)/(p3x*p1y - p2x*p1y - p3x*p0y + p2x*p0y - p3y*p1x + p2y*p1x + p3y*p0x - p2y*p0x);
    if(0 < s && s < 1){ // does not take the end points
        float t = (p2x + ((p3x - p2x) * s) - p0x)/(p1x - p0x);
        if(0 < t && t < 1){ // does not take the end points
             float pointX = p0x + ((p1x - p0x) * t);
             float pointY = p0y + ((p1y - p0y) * t);
             return ofVec2f(pointX, pointY, 0);
        }
    }
    return ofVec2f(-1.0f, 0.0f, 0); // if no intersection THEN returned point has x<0
}


__device__ ofVec2f segment::intersectionWithGivenSegment(segment other){
    // TODO: Implement A Fast Method To Find The Edge Intersection Point.
    // Should return the intersection point or null, if no intersection exists.
    //  Care should be taken to make the implementation CORRECT, but SPEED MATTERS.
     
//    segment other(ofVec2f(0.0f, 0.0f), ofVec2f(1500.0f, 0.0f)); // x axis as segment
    //do the p1 - p0 and use that so that number of multiplication is less
    float p0x = other.p0.x;
    float p0y = other.p0.y;
    float p1x = other.p1.x;
    float p1y = other.p1.y;
    float p2x = p0.x;
    float p2y = p0.y;
    float p3x = p1.x;
    float p3y = p1.y;
    float s = (p2y*p1x - p0y*p1x - p2y*p0x + p0y*p0x - p2x*p1y + p0x*p1y + p2x*p0y - p0x*p0y)/(p3x*p1y - p2x*p1y - p3x*p0y + p2x*p0y - p3y*p1x + p2y*p1x + p3y*p0x - p2y*p0x);
    if(0 < s && s < 1){ // does not take the end points
        float t = (p2x + ((p3x - p2x) * s) - p0x)/(p1x - p0x);
        if(0 < t && t < 1){ // does not take the end points
             float pointX = p0x + ((p1x - p0x) * t);
             float pointY = p0y + ((p1y - p0y) * t);
             return ofVec2f(pointX, pointY, 0);
        }
    }
    return ofVec2f(0.0f, 0.0f,0); // if no intersection THEN returned point has x<0, TODO how to pass null
}


__device__ bool segment::collinearWithQ(){
    return abs(p0.x * p1.y - p0.y * p1.x)/2;
}


vector<vray> segment::generateVray(segment seg){
    vector<vray> bothVray;
    ofVec2f xAxisVec(1.0,0.0);
    float infinity = 9999.0f;
    // printf("\n inside s.p0.x: %f, s.p0.y: %f, s.p1.x: %f, s.p1.y: %f", seg.p0.x, seg.p0.y, seg.p1.x, seg.p1.y);
    
    if((seg.p0.x * seg.p1.y - seg.p0.y * seg.p1.x) < 0){ // p0 X p1
        // to ensure p0 theta < p1 theta
        swap(seg.p0, seg.p1);
    }
    float theta0 = xAxisVec.angle(seg.p0);
    float theta1;
    if(xAxisVec.angle(seg.p1) == 0.0f){
        theta1 = 360.0;
    }
    else{
        theta1 = xAxisVec.angle(seg.p1);
    }
    theta0 = theta0 < 0? theta0 + 360.0f : theta0;
    theta1 = theta1 < 0? theta1 + 360.0f : theta1;
    // cout << "\n thetas" << theta0 << " " << theta1 ;
    bothVray.push_back(vray(theta0, seg.p0.getNormalized(), infinity, seg.p0.length()));
    bothVray.push_back(vray(theta1, seg.p1.getNormalized(), seg.p1.length(), infinity));
    
    return bothVray;
}

__device__ void mergeVraysGpu(vray *l1, vray *l2, vray *retVray, int mEach){
    float infinity = 99999.0;
    // vector<vray> l;
    // vray *l;
    // hipMalloc(&l, 10 * sizeof(vray));

    // ofVec2f ofV(0.0f, 0.0f, 0);
    // vray *l = {vray(0.0, ofV, 0.0, 0.0, 0)};
    // printf("\n l1[0].theta:%f, l2[0].theta:%f, m:%d", l1[0].theta, l2[0].theta, m);
    
    // int n = l1.size() + l2.size();
    // int n1 = l1.size();
    // int n2 = l2.size();
    int n = mEach * 2;
    int n1 = mEach;
    int n2 = mEach; //TODO check results
    
    // int i = 0; // diff from paper
    // int i1 = 0;
    // int i2 = 0;
    // int k = 0;
    // int t = 0;
    int i = 0; // diff from paper
    int i1 = 0;
    int i2 = 0;
    int k = 0;
    int t = 0;
    
    // vector<vray> lk;
    // vector<vray> lt;
    // int ik=0;
    // int it=0;
    vray *lk;
    vray *lt;
    int ik=0;
    int it=0;

    int pushCnt = 0;
    

    while(i < n){
        if(i2 >= n2){
            k = 1;
            ik = i1;
            lk = l1;
            t = 2;
            it = i2;
            lt = l2;
        }
        else if(i1 >= n1){
            k = 2;
            ik = i2;
            lk = l2;
            t = 1;
            it = i1;
            lt = l1;
        }
        else if(l1[i1].theta <= l2[i2].theta){
            k = 1;
            ik = i1;
            lk = l1;
            t = 2;
            it = i2;
            lt = l2;
        }
        else{
            k = 2;
            ik = i2;
            lk = l2;
            t = 1;
            it = i1;
            lt = l1;
        }

//         l.push_back(lk.at(ik));
        retVray[i] = lk[ik];
        pushCnt++;

        if(0 < it && it < mEach && lt[it].r < infinity ){
            float ox = lt[it-1].unitVec.x * lt[it-1].l;
            float oy = lt[it-1].unitVec.y * lt[it-1].l;
            ofVec2f o1(ox, oy, 0);
            ox = lt[it].unitVec.x * lt[it].r;
            oy = lt[it].unitVec.y * lt[it].r;
            ofVec2f o2(ox, oy, 0);
            segment s(o1, o2, 0);

            float thetaRad = retVray[i].theta * PI / 180.0;



            segment other(ofVec2f(0.0f, 0.0f, 0), ofVec2f(infinity * cos(thetaRad), infinity * sin(thetaRad), 0), 0); // x axis as segment
            ofVec2f p = s.intersectionWithGivenSegment(other);
           if(p.x == 0.0 && p.y == 0.0){
            //    cout << "\n point" << p.x << " " << p.y ;
               printf("\n it is null - did not find an intersection ");
           }
           else{
            //    cout << "\n it is null - did not find an intersection ";
                printf("\n point %f, %f", p.x, p.y);
           }
            

            retVray[i].l = min(retVray[i].l, p.lengthGpu());
            retVray[i].r = min(retVray[i].r, p.lengthGpu());
        }

        if(k == 1){
            i1 = i1 + 1; // can assign k = i1 or i2 initially based on condition so that we  have to chec value of k again n again
        }
        else{
            i2 = i2 + 1;
        }
        i = i + 1;
    }
    
    // l1 = retVray

    for(int i=0; i<mEach; i++){
        l1[i] = retVray[i];
    }

    // return l;
    return;
}

__global__ void preprocess_in_parallel(segment* input, int size, int x, int y, segment* output) {
 
    int tid = threadIdx.x;
    if(tid < size) {
        // ofVec2f* q = create_ofvec_obj(x, y);
        ofVec2f q(x, y, 0);
        // printf("Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
        input[tid].translateToQ(q);
        // printf("After translation to q Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
        if(input[tid].collinearWithQ() != 0.0f){
            if(input[tid].possibleIntersectionTestXAxis()) {
                ofVec2f splitPoint = input[tid].splitSegmentInto2();
                if(splitPoint.x != -1.0f){ 
                    // printf("splitting to q Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
                    output[2 * tid] = segment(input[tid].p0, splitPoint, 0);
                    output[2 * tid + 1] = segment(input[tid].p1, splitPoint, 0);
                    output[2 * tid].isValid = true;
                    output[2 * tid + 1].isValid = true;
                } else {
                    // printf("No splitting to q Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
                    output[2 * tid] = segment(input[tid].p0, input[tid].p1, 0);
                    output[2 * tid].isValid = true;
                }
            } else {
                output[2 * tid] = segment(input[tid].p0, input[tid].p1, 0);
                output[2 * tid].isValid = true;
                // printf("No splitting 2 to q Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
                // printf("No splitting 2 in output to q Id: %d, po.x: %f p1.x: %f\n", tid, output[2*tid].p0.x, output[2*tid].p1.x);
            }
        }
    }
}

__global__ void merge_in_parallel(vray *input_vrays, int size, vray *output_vrays){
    // shared memory 
    // extern __shared__ vray vrayData[];
  
    int tid = threadIdx.x;

    int n = 4;

    // if (tid < size){
    //     for(int m=4; m <= blockDim.x ; m = m * 2){
    //         printf("\n m:%d", m);
    //         if(tid%m == 0){ // merge 4 vrays here 
    //             printf("\n m=%d, tid:%d, it is vray0", m, tid);
    //             printf("\n \t %d, %d", tid, tid + (m/2));
    //             vray *v1 = &input_vrays[tid];
    //             vray *v2 = &input_vrays[tid+(m/2)];
    //             vray *res = &output_vrays[tid];
    //             // mergeVraysGpu(v1, v2, res, m);

    //         }
    //         // else{
    //             // printf("\n m=%d, tid:%d, it is other vray", m, tid);
    //             // do nothing 
    //         // }

    //         __syncthreads();
    //         //TODO kill the thread not required?
    //     }
    // }

     if (tid < size){
        for(int m=4; m < 2*size ; m = m * 2){ 
            printf("\n m:%d", m);
            if(tid%m == 0){ // merge 4 vrays here 
                printf("\n m=%d, tid:%d, it is vray0", m, tid);
                printf("\n \t %d, %d", tid, tid + (m/2));
                vray *v1 = &input_vrays[tid];
                vray *v2 = &input_vrays[tid+(m/2)];
                vray *res = &output_vrays[tid];

                mergeVraysGpu(v1, v2, res, m);

            }

            __syncthreads();
            //TODO kill the thread not required?
        }
    }

}

__global__ void generate_vrays_in_parallel(segment* input, int size, vray* output) {
 
    int tid = threadIdx.x;
    if(tid < size) {
        printf("Id: %d, po.x: %f p1.x: %f\n", tid, input[tid].p0.x, input[tid].p1.x);
        // vector<vray> segVrays = input[tid].generateVray(input[tid]);
        // output[2 * tid] = segVrays.at(0);
        // output[2 * tid + 1] = segVrays.at(1);
    }
}

class GPU_V1 {
    float infinity = 9999.0f;

    public:

    vector<segment> preprocess(vector<segment>& segments, ofVec2f& q) {
            
        hipError_t err = hipSuccess;

        segment *d_segments;
        hipMalloc(&d_segments, segments.size() * sizeof(segment));

        err = hipMemcpy(d_segments, segments.data(), segments.size() * sizeof(segment), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate d_segments (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        segment *d_output_segments;
        hipMalloc(&d_output_segments, 2 * segments.size() * sizeof(segment));

        int threadsPerBlock = 100;
        int blocksPerGrid = 1;
        preprocess_in_parallel <<<blocksPerGrid, threadsPerBlock>>> (d_segments, segments.size(), q.x, q.y, d_output_segments);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch preprocess_in_parallel kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        segment* preprocessed_segments = (segment*) malloc(sizeof(segment) * 2 * segments.size());
        err = hipMemcpy(preprocessed_segments, d_output_segments, 2 * segments.size() * sizeof(segment), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy d_output_segments kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        vector<segment> filtered_segments;
        for(int i=0;i<2*segments.size();i++) {
            if(preprocessed_segments[i].isValid) {
                filtered_segments.push_back(preprocessed_segments[i]);
            }
        }
        // hipFree(d_output_segments);
        // hipFree(d_segments);
        return filtered_segments;

    }

    vector<vray> generate_vrays_from_segments(vector<segment>& segments) {
            
        hipError_t err = hipSuccess;

        segment *d_segments;
        hipMalloc(&d_segments, segments.size() * sizeof(segment));

        err = hipMemcpy(d_segments, segments.data(), segments.size() * sizeof(segment), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate d_segments (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        vray *d_output_vrays;
        hipMalloc(&d_output_vrays, 2 * segments.size() * sizeof(vray));

        int threadsPerBlock = 100;
        int blocksPerGrid = 1;
        generate_vrays_in_parallel <<<blocksPerGrid, threadsPerBlock>>> (d_segments, segments.size(), d_output_vrays);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch generate_vrays_in_parallel kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        vray* gpu_generated_vrays = (vray*) malloc(sizeof(vray) * 2 * segments.size());
        err = hipMemcpy(gpu_generated_vrays, d_output_vrays, 2 * segments.size() * sizeof(vray), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy d_output_vrays kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        vector<vray> generated_vrays;
        for(int i=0;i<2*segments.size();i++) {
            cout << " " << gpu_generated_vrays[i].theta ;
                generated_vrays.push_back(gpu_generated_vrays[i]);
        }
        hipFree(d_output_vrays);
        hipFree(d_segments);
        return generated_vrays;

    }

    vray* mergeVraysGpu(vector<vray>& initial_vray_list){
        cout << "\n do";
        vector<vray> ret;

  
        // hipError_t err = hipSuccess;

        // segment *d_segments;
        // hipMalloc(&d_segments, segments.size() * sizeof(segment));

        // err = hipMemcpy(d_segments, segments.data(), segments.size() * sizeof(segment), hipMemcpyHostToDevice);
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to allocate d_segments (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        hipError_t err = hipSuccess;

        vray *d_vrays;
        hipMalloc(&d_vrays, initial_vray_list.size() * sizeof(vray));

        err = hipMemcpy(d_vrays, initial_vray_list.data(), initial_vray_list.size() * sizeof(vray), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate d_vrays (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // segment *d_output_segments;
        // hipMalloc(&d_output_segments, 2 * segments.size() * sizeof(segment));

        vray *d_output_vrays;
        hipMalloc(&d_output_vrays, initial_vray_list.size() * sizeof(vray));

        // int threadsPerBlock = 100;
        // int blocksPerGrid = 1;
        int threadsPerBlock = 16;
        int blocksPerGrid = 1;
        merge_in_parallel <<<blocksPerGrid, threadsPerBlock>>> (d_vrays, initial_vray_list.size(), d_output_vrays);
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch preprocess_in_parallel kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        vray* return_vrays = (vray*) malloc(sizeof(vray) * initial_vray_list.size());
        err = hipMemcpy(return_vrays, d_output_vrays, initial_vray_list.size() * sizeof(vray), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy d_output_segments kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        // vector<segment> filtered_segments;
        // for(int i=0;i<2*segments.size();i++) {
        //     if(preprocessed_segments[i].isValid) {
        //         filtered_segments.push_back(preprocessed_segments[i]);
        //     }
        // }
        hipFree(d_output_vrays);
        hipFree(d_vrays);
        return return_vrays;

    }

    virtual vector<vray> process_segments(vector<segment> segments, ofVec2f& q) {
        // start = 0;
        std::cout<<"\n Starting Preprocess, size: "<< segments.size() <<std::endl;
        vector<segment> updated_segments = this->preprocess(segments, q);
        std::cout<<"\n Preprocess complete, size: "<< updated_segments.size() <<std::endl;

        // for(int i=0; i<updated_segments.size(); i++){
        //     printf("Id: %d, p0.x: %f p0.y: %f, p1.x: %fp1.y: %f\n", i, updated_segments.at(i).p0.x, updated_segments.at(i).p0.y, updated_segments.at(i).p1.x, updated_segments.at(i).p1.y);
        // }

        std::cout<<"\n Starting generate vrays"<<std::endl;
        vector<vray> initial_vrays;// = this->generate_vrays_from_segments(updated_segments);
        for(int i =0; i<updated_segments.size(); i++){
            segment s = updated_segments.at(i);
            // printf("i: %d, s.p0.x: %f, s.p0.y: %f, s.p1.x: %f, s.p1.y: %f\n", i, s.p0.x, s.p0.y, s.p1.x, s.p1.y);
            initial_vrays.push_back(s.generateVray(s).at(0));
            initial_vrays.push_back(s.generateVray(s).at(1));
            // printf("i: %d, .theta: %f\n", i, s.generateVray(s).at(0).theta);
        }
        std::cout<<"\n End of generate vrays, size: "<< initial_vrays.size() << std::endl;
        // for(int i=0; i<initial_vrays.size(); i++){
        //     printf("i: %d, .theta: %f\n", i, initial_vrays.at(i).theta);
        // }

        vector<vray> merged_vrays;
        std::cout<<"\n Starting merge vrays"<<std::endl;
        vray *merged_vrayssss = this->mergeVraysGpu(initial_vrays);
        std::cout<<"\n Ending merge"<<std::endl;
        // for(int i=0; i<initial_vrays.size(); i++){
        //     printf("i: %d, .theta: %f\n", i, merged_vrayssss[i].theta);
        // }
        // end = 0;
        return merged_vrays;
    }
};

vector<segment> generateSegments(int n){
    vector<segment> ret;
    // int full = n%
    float r = 5.0f;
    float theta = 0.1f;
    for (int i=0; i<n; i++) {
        r = r + 3.0;
        theta = theta + 0.1;
        float x0 = r * cos(theta) + 500;
        float y0 = r * sin(theta) + 400;
        r = r + 3.0;
        theta = theta + 0.1;
        float x1 = r * cos(theta)+ 500;
        float y1 = r * sin(theta)+ 400;
        cout << x0 << y0 << y0 << y1<< endl;
        ret.push_back(segment(ofVec2f(x0, y0), ofVec2f(x1, y1)));
    }
    return ret;
}

void test_preprocessing() {

    vector<int> all_time;
    int iteration = 500;
    double timeDuration = 0.0;

    
    vector<segment> listSegments = generateSegments(iteration);
    ofVec2f pointQ(500,400);

    GPU_V1 *gpu = new GPU_V1();
    int iteration = 500;
    double timeCpu = 0.00;
    vector<segment> updated_segments;
    for(int i=0; i<iteration; i++){
        time_t start = time(NULL);
        updated_segments = gpu->preprocess(listSegments, q);
        time_t end = time(NULL);
        // double duration = double(end-begin) / CLOCKS_PER_SEC;
        double duration = double(end-start);
        timeCpu = timeCpu + duration;
    }
    cout.precision(17);
    cout << fixed << "\n Preprocessing in GPU: " << timeCpu/iteration;

    vector<vray> initial_vrays;
    for(int i =0; i<updated_segments.size(); i++){
        segment s = updated_segments.at(i);
        initial_vrays.push_back(s.generateVray(s).at(0));
        initial_vrays.push_back(s.generateVray(s).at(1));
    }
    
    timeCpu = 0.00;
    for(int i=0; i<iteration; i++){
        time_t start = time(NULL);

        gpu->mergeVraysGpu(initial_vrays);

        time_t end = time(NULL);
        // double duration = double(end-begin) / CLOCKS_PER_SEC;
        double duration = double(end-start);
        timeCpu = timeCpu + duration;
    }
    cout.precision(17);
    cout << fixed << "\n Merge Sequential time in GPU: " << timeCpu/iteration;
    
    // for(int i=0;i<iteration;i++) {
    // //     vector<segment> listSegments = {
    // //         segment(ofVec2f(600.0f, 550.0f), ofVec2f(650.0f, 400.0f)),  // right small, line to split at 0 degree
    // //         segment(ofVec2f(100.0f, 500.0f), ofVec2f(350.0f, 300.0f)),  // connected pair above
    // //         segment(ofVec2f(550.0f, 700.0f), ofVec2f(100.0f, 500.0f)),  // connected pair bottom
    // //         segment(ofVec2f(600.0f, 600.0f), ofVec2f(800.0f, 150.0f)),  // right side long
    // //         segment(ofVec2f(200.0f, 300.0f), ofVec2f(700.0f, 100.0f)),  // top horizontal

    // // //        segment(ofVec2f(800.0f, 500.0f), ofVec2f(800.0f, 500.01f)),   // the line for ending
    // //         segment(ofVec2f(450.0f, 450.0f), ofVec2f(400.0f, 400.0f))   // the collinear line
    // //     };
        
        

        
    //     // auto start = chrono::high_resolution_clock::now();
    //     // std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();
    //     clock_t begin = clock();

    //     vector<segment> segments = gpu->preprocess(listSegments, pointQ);
    //     // auto stop = chrono::high_resolution_clock::now();
    //     // std::chrono::steady_clock::time_point stop = std::chrono::steady_clock::now();
    //     clock_t end = clock();
    //     double duration = double(end-begin) / CLOCKS_PER_SEC;
    //     // auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    //     // all_time.push_back(duration);
    //     timeDuration = timeDuration + duration;
    // }
    //return timeDuration/iteration;//(float)std::reduce(all_time.begin(), all_time.end())/(float) all_time.size();
    // cout << "\n Prreprocessing in GPU: " << timeDuration/iteration;
}

int main() {

    //TEST Preprocessing
    // std::cout<<"Time taken For Preprocessing in GPU: ";//<< test_preprocessing() <<std::endl;
    // test_preprocessing();

    vector<segment> listSegments = {
        segment(ofVec2f(600.0f, 550.0f), ofVec2f(650.0f, 400.0f)),  // right small, line to split at 0 degree
        segment(ofVec2f(100.0f, 500.0f), ofVec2f(350.0f, 300.0f)),  // connected pair above
        segment(ofVec2f(550.0f, 700.0f), ofVec2f(100.0f, 500.0f)),  // connected pair bottom
        segment(ofVec2f(600.0f, 600.0f), ofVec2f(800.0f, 150.0f)),  // right side long
        segment(ofVec2f(200.0f, 300.0f), ofVec2f(700.0f, 100.0f)),  // top horizontal

//        segment(ofVec2f(800.0f, 500.0f), ofVec2f(800.0f, 500.01f)),   // the line for ending
        segment(ofVec2f(450.0f, 450.0f), ofVec2f(400.0f, 400.0f))   // the collinear line
    };
    
    ofVec2f pointQ(500,400);
    GPU_V1 *v1 = new GPU_V1();
    v1->process_segments(listSegments, pointQ);
    return 0;
}

